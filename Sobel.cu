#include "hip/hip_runtime.h"
//
// Created by wyl on 2021/9/17.
//
#include<iostream>
#include <opencv2/opencv.hpp>
using namespace std;
using namespace cv;
__global__ void SobelInCuda(char* src, char* dst, int width, int height){
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;
    int index = idy * width + idx;
    int gx, gy;
    if(idx>0 && idx<width-1 && idy>0 && idy<height-1){
        gx = -src[index-width-1]+src[index-width+1]-2*src[index-1]+2*src[index+1]-src[index+width-1]+src[index+width+1];
        gy = -src[index-width-1]-2*src[index-width]-src[index-width+1]+src[index+width-1]+2*src[index+width]+src[index+width+1];
        dst[index] = (abs(gx) + abs(gy)) / 2;
    }
}

int main(){
    Mat img = imread("../1.jpg", 0);
    int width = img.cols;
    int height = img.rows;

    // 创建GPU内存
    char *dst;
    char *src;
    hipMalloc((void**)&dst, width*height*sizeof(char));
    hipMalloc((void**)&src, width*height*sizeof(char));

    // 定义grid和block
    dim3 blockSize(32, 32);
    dim3 gridSize(width/blockSize.x+1, height/blockSize.y+1);

    // img传递到gpu
    hipMemcpy(src, img.data, width*height*sizeof(char), hipMemcpyHostToDevice);

    // 运行核函数
    SobelInCuda<<<gridSize, blockSize>>>(src, dst, width, height);

    // gpu返回至cpu
    Mat res(height, width, CV_8UC1, Scalar(0));
    hipMemcpy(res.data, dst, width*height*sizeof(char), hipMemcpyDeviceToHost);
    hipFree(src);
    hipFree(dst);
    imshow("", res);
    waitKey();
};