#include "hip/hip_runtime.h"
//
// Created by wyl on 2021/9/17.
//
#include<iostream>
#include <opencv2/opencv.hpp>
using namespace std;
using namespace cv;

#define byte unsigned char
// 有问题
__global__ void SobelInCuda(const byte* src, byte* dst, const int width, const int height){
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;
    int index = idy * width + idx;
    float gx, gy;
    if(idx>0 && idx<width-1 && idy>0 && idy<height-1){
        gx = -src[index-width-1]+src[index-width+1]-2*src[index-1]+2*src[index+1]-src[index+width-1]+src[index+width+1];
        gy = -src[index-width-1]-2*src[index-width]-src[index-width+1]+src[index+width-1]+2*src[index+width]+src[index+width+1];
        dst[index] = (abs(gx) + abs(gy)) / 2;
//        dst[index] = sqrt(gx*gx + gy*gy);
    }
}

int main(){
    Mat img = imread("../1.jpg", 0);
    int width = img.cols;
    int height = img.rows;

    int len = width * height * sizeof(byte);

    byte* imgData = new byte[len];
    std::memcpy(imgData, img.data, len);

    // 创建GPU内存
    byte *dst;
    byte *src;
    hipMalloc((void**)&dst, width*height*sizeof(byte));
    hipMalloc((void**)&src, width*height*sizeof(byte));

    // img传递到gpu
    hipMemcpy(src, imgData, width*height*sizeof(byte), hipMemcpyHostToDevice);

    // 定义grid和block
    dim3 blockSize(32, 32);
    dim3 gridSize(width/blockSize.x+1, height/blockSize.y+1);

    // 运行核函数
    SobelInCuda<<<gridSize, blockSize>>>(src, dst, width, height);

    // gpu返回至cpu
    hipMemcpy(imgData, dst, width*height*sizeof(byte), hipMemcpyDeviceToHost);

    Mat res(height, width, CV_8UC1, imgData);

    hipFree(src);
    hipFree(dst);
    imshow("cuda", res);
    imwrite("cuda.jpg", res);
    waitKey();
}
