//
// Created by wyl on 2020/11/14.
//

#include <iostream>
#include <sys/time.h>
#include <math.h>
#include <hip/hip_runtime.h>


using namespace std;

// 核函数
// 1D1D  两个向量加法kernel
__global__ void add(float* x, float * y, float* z, int n)
{
    // 获取全局索引
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    z[index] = x[index] + y[index];
}

int main()
{
    int N = 102400;
    int nBytes = N * sizeof(float);

    // 申请托管内存
    float *x, *y, *z;
    hipMallocManaged((void**)&x, nBytes);
    hipMallocManaged((void**)&y, nBytes);
    hipMallocManaged((void**)&z, nBytes);

    // 初始化数据
    for (int i = 0; i < N; ++i)
    {
        x[i] = 10.0;
        y[i] = 20.0;
    }

    // 定义kernel的执行配置
    dim3 blockSize(512);
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x);
    // 执行kernel
    // 创建gridSize个线程块在GPU上运行
    for(int i=0; i<1000; i++){
        add <<< gridSize, blockSize >>>(x, y, z, N);
    }

    // 同步device 保证结果能正确访问
    hipDeviceSynchronize();
    // 检查执行结果
    float maxError = 0.0;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(z[i] - 30.0));
    std::cout << "最大误差: " << maxError << std::endl;

    // 释放内存
    hipFree(x);
    hipFree(y);
    hipFree(z);

    return 0;
}