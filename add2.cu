//
// Created by wyl on 2022/2/4.
//

#include <hip/hip_runtime.h>
#include <iostream>

__global__ void add(int* a, int* b, int* c, int num){
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index >= 0 && index < num){
        c[index] = a[index] + b[index];
    }
}

int main(){
    int num = 10000;
    int* a = new int[num];
    int* b = new int[num];
    for(int i=0; i<num; i++){
        a[i] = i;
        b[i] = i*i;
    }

    int *a_cuda, *b_cuda, *c_cuda;
    hipMalloc((void**)&a_cuda, num * sizeof(int));
    hipMalloc((void**)&b_cuda, num * sizeof(int));
    hipMalloc((void**)&c_cuda, num * sizeof(int));

    hipMemcpy(a_cuda, a, num*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(b_cuda, b, num*sizeof(int), hipMemcpyHostToDevice);

    dim3 blockSize(256);
    dim3 gridSize(num/blockSize.x+1);
    add<<<gridSize, blockSize>>>(a_cuda, b_cuda, c_cuda, num);

    int* c = new int[num];
    hipMemcpy(c, c_cuda, num*sizeof(int), hipMemcpyDeviceToHost);

    // cpu
    int* c_cpu = new int[num];
    for(int i=0; i<num; i++){
        c_cpu[i] = a[i] + b[i];
    }

    for(int i=0; i<num; i++){
        printf("%d + %d = %d vs %d\n", (int)a[i], (int)b[i], (int)c[i], (int)c_cpu[i]);
    }

    hipFree(a_cuda);
    hipFree(b_cuda);
    hipFree(c_cuda);

    delete[] a;
    delete[] b;
    delete[] c;

    return 0;

}
